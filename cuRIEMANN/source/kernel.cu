#include "hip/hip_runtime.h"
#include <stdio.h>

#include <math.h>
#include <new> //std::nothrow

#include "Complex.h"

#include "kernel.h"

int const threadCount = 192; //Number of threads per block

//Dimensions
unsigned globalWidth;
unsigned globalHeight;

//Flags
bool isHost; //host = true, false = device
bool isShared; //shared memory = true, global memory = false

bool isVerbose; //For debugging purposes

//Arrays
Token* globalList;
RGB* globalDeviceResults;
RGB* globalHostResults;
Complex* globalStack;
Complex* globalDoubleStack;

unsigned globalListCount;
unsigned globalMaxStack;

//Array sizes
unsigned resultsSize;
unsigned stackSize;

__host__ ERRORCODES entryConstruct(int flags = 0)
{
	int deviceCount;

	isVerbose = (flags & 0x1);

	printf("Flag list:\n");
	printf("Verbose: ");
	if (flags & 0x1)
		printf("On\n");
	else
		printf("Off\n");

	printf("Force Host: ");
	if (flags & 0x2)
		printf("On\n");
	else
		printf("Off\n");

	if (isVerbose)
		printf("Entry Construct\n");

	hipError_t firstCall = hipGetDeviceCount(&deviceCount); 

	if (flags & 0x2) //Force host
		isHost = true;
	else  //Try device, otherwise use host
		isHost = firstCall == hipErrorNoDevice || firstCall == hipErrorInsufficientDriver;//No device or no proper device drivers, use host

	if (!isHost)  //Use Device
	{
		int device;
		for (device = 0; device < deviceCount; ++device)
		{
			hipDeviceProp_t deviceProp;
			hipGetDeviceProperties(&deviceProp, device);
			if (isVerbose)
				printf("	Device %d has compute capability %d.%d.\n", device, deviceProp.major, deviceProp.minor);
		}

		{
			int* a;
			if (hipMalloc(&a, sizeof(int)) == hipErrorOutOfMemory)
				return Construct_Test_MemAlloc;
			if (hipFree(a) == hipErrorInvalidDevicePointer)
				return Construct_Test_InvalidPointer;
		}
		if (isVerbose)
			printf("	Using device...\n");

	}
	else //Otherwise use host
	{
		if (isVerbose)
			printf("	Using Host...\n");
	}
	
	

	globalList = nullptr;
	globalDeviceResults = nullptr;
	globalHostResults = nullptr;
	globalStack = nullptr;
	globalDoubleStack = nullptr;

	stackSize = 0;
	resultsSize = 0;
	return Success;
}

__host__ ERRORCODES entryInitialise(unsigned width, unsigned height, TokenList list, RGB* results)
{
	globalMaxStack = list.getStackSize();

	if (isVerbose)
		printf("Entry Initialise\n");

	if (isHost) //Uses - results, d_stack, globalList
	{
		if (stackSize < width * height * sizeof(Complex) * globalMaxStack) //Current stack array is too small, reallocate
		{
			if (isVerbose)
				printf("	Reallocate host stack - Host\n");
			//delete[] globalStack; //Delete current stack array if allocated
			globalStack = new (std::nothrow) Complex[width*height*globalMaxStack]; //Construct new stack array
			if (globalStack == nullptr)
				return Initialise_Host_Stack_MemAlloc;
			stackSize = width * height * globalMaxStack * sizeof(Complex); //Update stackSize
		}
		//delete[] globalList; //free current token list
		globalList = list.formula(); //Update new tokenlist
	}
	else //Uses - globalList, d_results, results and d_stack
	{
		if (resultsSize < width * height * sizeof(Complex)) //Current results array is too small, reallocate
		{
			if (isVerbose)
			{
				printf("	Reallocate device results - Device - old size: %i, new size: %i\n", resultsSize, width * height * sizeof(Complex));
				printf("	width: %i, height: %i, sizeof: %i\n", width, height, sizeof(Complex));
			}
			if (hipFree(globalDeviceResults) == hipErrorInvalidDevicePointer) //Delete current device results array if allocated
				return Initialise_Device_DeviceResults_InvalidPointer;
			if (hipMalloc(&globalDeviceResults, width * height * sizeof(RGB)) == hipErrorOutOfMemory) //Construct new device array
				return Initialise_Device_DeviceResults_MemAlloc;
			resultsSize = width * height * sizeof(RGB); //Update resultSize
		}

		if (isVerbose)
			printf("	Stack max: %i\n", globalMaxStack);
		isShared = globalMaxStack < 5;

		if (!isShared && stackSize < width * height * sizeof(Complex) * globalMaxStack) //If we're using global memory and the memory needs expanding...
		{
			if (isVerbose)
				printf("	Reallocate device stack - device\n");
			if (hipFree(globalStack) == hipErrorInvalidDevicePointer) //Delete current stack array if allocated
				return Initialise_Device_Stack_InvalidPointer;
			if (hipMalloc(&globalStack, width * height * globalMaxStack * sizeof(Complex)) == hipErrorOutOfMemory) //Construct new stack array
				return Initialise_Device_Stack_MemAlloc;
			stackSize = width * height * globalMaxStack * sizeof(Complex); //Update stackSize
		}
		if (isVerbose)
			printf("	Copying list to device\n");
		if (hipFree(globalList) == hipErrorInvalidDevicePointer) //Free device token list
			return Initialise_Device_List_InvalidPointer;
		if (hipMalloc(&globalList, list.count() * sizeof(Token)) == hipErrorOutOfMemory) //Allocate new token list
			return Initialise_Device_List_MemAlloc;
		if (hipMemcpy(globalList, list.formula(), list.count() * sizeof(Token), hipMemcpyHostToDevice) != hipSuccess) //Copy from host to device
			return Initialise_Copy_List_Error;
	}
	//Update new width and height
	globalWidth = width;
	globalHeight = height;
	globalListCount = list.count();
	globalHostResults = results;
	return Success;
}

__host__ ERRORCODES entryCalculate(Complex min, Complex max)
{
	if (isVerbose)
		printf("Entry Calculate\n");

	int N = globalWidth * globalHeight;
	ERRORCODES err = Success;

	auto first = Complex(min.real(), max.imag());

	auto last = Complex(max.real(), min.imag());

	if (!isHost)
	{
		if (isVerbose)
			printf("	Single-precision Kernel Execuation\n");

		if (isShared)
			sharedCalculatef << < N / threadCount + (N % threadCount ? 1 : 0), threadCount, threadCount * globalMaxStack * sizeof(Complex) >> >(globalDeviceResults, globalList, globalListCount, globalMaxStack, first, last - first, globalWidth, globalHeight, globalWidth * globalHeight);
		else
			globalCalculate << < N / threadCount + (N % threadCount ? 1 : 0), threadCount >> >(globalDeviceResults, globalStack, globalList, globalListCount, globalMaxStack, first, last - first, globalWidth, globalHeight, globalWidth * globalHeight);
		
		if (hipMemcpy(globalHostResults, globalDeviceResults, N*sizeof(RGB), hipMemcpyDeviceToHost) != hipSuccess)
			err = Calculate_Copy_Results_Error;
		else
			err = Success;

	}
	else
	{
		hostCalculate(globalHostResults, globalStack, globalList, globalListCount, globalMaxStack, first, last - first, globalWidth, globalHeight);
	}

	return err;
}

__host__ ERRORCODES entryDestruct()
{
	if (isVerbose)
		printf("Entry Destruct\n");

	ERRORCODES err = Success;
	if (isHost)
	{
		//delete[] globalStack;
	}
	else
	{
		if (hipFree(globalList) == hipErrorInvalidDevicePointer)
			err = Destruct_Device_List_InvalidPointer;
		if (hipFree(globalDeviceResults) == hipErrorInvalidDevicePointer)
			err = Destruct_Device_DeviceResults_InvalidPointer;
		if (hipFree(globalStack) == hipErrorInvalidDevicePointer)
			err = Destruct_Device_Stack_InvalidPointer;
	}
	return err;
}

__host__ void entryTranslate()
{

}

__host__ void entryTrace(Complex variable, TokenList list, Complex *ans, RGB *col = nullptr, double* mod = nullptr, double* arg = nullptr)
{
	auto stackmax = list.getStackSize();
	Complex* stack = new Complex[stackmax];

	--stack; //Stack is a before the beginning pointer, so decrement before use

	*ans = calculate(variable, list.formula(), list.count(), stack, 1);

	if (mod != nullptr)
		*mod = thrust::abs(*ans);
	if (arg != nullptr)
		*arg = thrust::arg(*ans);
	if (col != nullptr)
		*col = color(*ans);

	//delete stack;
}

/*__host__ Complex entryGradient(TokenList list, Complex value)
{
	return Complex();
}*/

__host__ Complex entryNewtonRaphson(TokenList list, Complex xn, int timeout)
{
	if (timeout-- == 0)
		return xn;

	auto next = xn - traceWrapper(list, xn) / fast_gradient(list, xn);

	if (thrust::abs(next - xn) < MAGIC)
		return next;

	return entryNewtonRaphson(list, next, timeout);
}

int main()
{


	entryConstruct(true);

	auto answer = new RGB[1920*1080];

	TokenList list;
	unsigned count = 3;

	Token* tokens = new Token[count];

	tokens[0] = { 1, { 0, 0 } };
	tokens[1] = { 1, { 0, 0 } };
	tokens[2] = { 2, { 4, 0 } };

	list = { count, tokens };

	entryInitialise(1920, 1080, list, answer); //1920 x 1080 f(z) = z Quickest, simplest full HD graph

	entryCalculate({ -200.0f, -200.0f }, { 300.0f, 300.0f });

	RGB color; 
	thrust::complex<double> z;
	double mod, arg;

	entryTrace({ 3.0, 0.0 }, list, &z, &color, &mod, &arg);

	printf("answer: %f, %f - %f, %f\n", z.real(), z.imag(), mod, arg);

	//entryInitialise(1920, 1080, TokenList<Precision>({ { 1, { 0, 0 } }, { 1, { 1, 0 } }, { 2, { 4, 0 } } })); //1920 x 1080 f(z) = z ^ z Binary operation full HD graph

	//answer = entryCalculate({ -2.0f, -2.0f }, { 2.0f, 2.0f }, error);
	
	//entryInitialise(1920, 1080, TokenList<Precision>({ { 1, { 0, 0 } }, { 1, { 0, 0 } }, { 1, { 0, 0 } }, { 1, { 0, 0 } }, { 1, { 0, 0 } }, { 1, { 0, 0 } }, { 2, { 1, 0 } }, { 2, { 1, 0 } }, { 2, { 1, 0 } }, { 2, { 1, 0 } }, { 2, { 1, 0 } } })); //1920 x 1080 f(z) = z + z + z + z + Z Tests globalCalculate kernel

	//entryCalculate({ -2.0f, -2.0f }, { 2.0f, 2.0f }, error);
	
	//entryInitialise(1920 * 2, 1080 * 2, TokenList<Precision>({ { 3, { 1, 0 } }, { 1, { 0, 0 } }, { 1, { 0, 0 } }, { 2, { 2, 0 } }, { 2, { 1, 0 } }, { 2, { 9, 0 } } })); //4k f(z) = ln(z^2 + 1) //Somewhat difficult 4K graph

	//answer = entryCalculate({ -2.0f, -2.0f }, { 2.0f, 2.0f }, error);

	//entryInitialise(1920 * 2, 1080 * 2, TokenList<Precision>({ { 1, { 0, 0 } }, { 3, { 1, 0 } }, { 2, { 4, 0 } }, { 2, { 23, 0 } }, { 1, { 0, 0 } }, { 3, { 2, 0 } }, { 2, { 4, 0 } }, { 2, { 23, 0 } }, { 2, { 1, 0 } } })); //4k First few terms of riemann zeta function //Somewhat difficult 4K graph
	
	//answer = entryCalculate({ -2.0f, -2.0f }, { 2.0f, 2.0f }, error);

	//entryInitialise(1920, 1080, TokenList<Precision>({ { 1, { 0, 0 } }, { 1, { 0, 0 } }, { 2, { 4, 0 } } })); //1920 x 1080 f(z) = ln(z) Unary operation full HD graph

	//answer = entryCalculate({ 2.0f, 2.0f }, { 1002.0f, 1002.0f }, error);

	for (int i = 0; i < 10; i++)
		printf("%i %i %i %i - ind: %i\n", (int)answer[i].a, (int)answer[i].r, (int)answer[i].g, (int)answer[i].b, i);


	//for (int i = 1920 * 1080 - 10; i < 1920 * 1080; i++)
	//	printf("(%i) %i %i %i - ind: %i\n", (int)answer[i].a, (int)answer[i].r, (int)answer[i].g, (int)answer[i].b, i);

	hipDeviceReset();

	while (1);
}